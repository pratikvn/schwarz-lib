#include <hip/hip_runtime.h>

#define BLOCK_SIZE 512

template <typename ValueType, typename IndexType>
__global__ void
gather_kernel(const IndexType num_elems, const IndexType *indices,
              const ValueType *gather_from, ValueType *gather_into) {
  int row = blockDim.x * blockIdx.x + threadIdx.x;
  if (row < num_elems) {
    gather_into[row] = gather_from[indices[row]];
  }
}

template <typename ValueType, typename IndexType>
__global__ void
scatter_kernel(const IndexType num_elems, const IndexType *indices,
               const ValueType *scatter_from, ValueType *scatter_into) {
  int row = blockDim.x * blockIdx.x + threadIdx.x;
  if (row < num_elems) {
    scatter_into[indices[row]] = scatter_from[row];
  }
}

template <typename ValueType, typename IndexType>
void gather_values(const IndexType num_elems, const IndexType *indices,
                   const ValueType *gather_from, ValueType *gather_into) {
  dim3 grid((num_elems + BLOCK_SIZE - 1) / BLOCK_SIZE, 1, 1);

  gather_kernel<<<grid, BLOCK_SIZE, 0, 0>>>(num_elems, indices, gather_from,
                                            gather_into);
}

template <typename ValueType, typename IndexType>
void scatter_values(const IndexType num_elems, const IndexType *indices,
                    const ValueType *scatter_from, ValueType *scatter_into) {
  dim3 grid((num_elems + BLOCK_SIZE - 1) / BLOCK_SIZE, 1, 1);

  scatter_kernel<<<grid, BLOCK_SIZE, 0, 0>>>(num_elems, indices, scatter_from,
                                             scatter_into);
}

#define INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(_macro)                      \
  template _macro(float, int);                                                 \
  template _macro(double, int);                                                \
  template _macro(int, int);                                                   \
  template _macro(long int, int);                                              \
  template _macro(float, long int);                                            \
  template _macro(double, long int);                                           \
  template _macro(int, long int);                                              \
  template _macro(long int, long int);

#define DECLARE_GATHER(ValueType, IndexType)                                   \
  void gather_values(const IndexType, const IndexType *, const ValueType *,    \
                     ValueType *)
INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(DECLARE_GATHER);
#undef DECLARE_GATHER

#define DECLARE_SCATTER(ValueType, IndexType)                                  \
  void scatter_values(const IndexType, const IndexType *, const ValueType *,   \
                      ValueType *)
INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(DECLARE_SCATTER);
#undef DECLARE_SCATTER